#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/cuda/detail/IndexUtils.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include "ATen/TensorUtils.h"
// #include "ATen/Type.h"
#include "ATen/AccumulateType.h"
#include <THC/THCGeneral.h>
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

#include "type_shim.h"

typedef enum{
    ADAM_MODE_0   =0, // eps under square root
    ADAM_MODE_1   =1  // eps outside square root
} adamMode_t;

template <typename GRAD_T>
__global__ void strided_check_finite_cuda_kernel(
        volatile int* noop_gmem,
        GRAD_T* __restrict__ p_copy,
        const size_t tsize,
        int stride,
        int clear_overflow_first)
{
    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock) * stride;
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock*stride;

    if (clear_overflow_first) {
        if (i == 0) {
            *noop_gmem = 0;
        }
        __syncthreads();
    }

    for (int j = i; j < tsize; j+=totThreads) {
        GRAD_T pi = p_copy[j];
        if (!isfinite(pi)) {
            *noop_gmem = 1;
        }
    }
}

template <typename T, typename GRAD_T>
__global__ void adam_cuda_kernel(
        T* __restrict__ p,
        GRAD_T* __restrict__ p_copy, // For mixed precision training, pass NULL if not needed
        T* __restrict__ m,
        T* __restrict__ v,
        const GRAD_T * __restrict__ g,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        const size_t tsize,
        adamMode_t mode,
        const float decay)
{
    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock);
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock;

    T mi[ILP];
    T vi[ILP];
    T pi[ILP];
    T gi[ILP];

    bool overflow = false;
    for(int j_start = 0;  j_start < tsize;  j_start+=totThreads*ILP) {
#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            mi[ii] = T(0);
            vi[ii] = T(0);
            pi[ii] = T(0);
            gi[ii] = GRAD_T(0);

            int j = j_start + i + totThreads*ii;
            if (j < tsize) {
                pi[ii] = p[j];
                mi[ii] = m[j];
                vi[ii] = v[j];
                gi[ii] = static_cast<T>(g[j]);
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            T scaled_grad = gi[ii]/grad_scale;
            if (isfinite(scaled_grad)) {
                mi[ii] = b1*mi[ii] + (1-b1)*scaled_grad;
                vi[ii] = b2*vi[ii] + (1-b2)*scaled_grad*scaled_grad;
                float denom;
                if (mode == ADAM_MODE_0)
                    denom = sqrtf(vi[ii] + eps);
                else // Mode 1
                    denom = sqrtf(vi[ii]) + eps;
                float update = (mi[ii]/denom) + (decay*pi[ii]);
                pi[ii] = pi[ii] - (step_size*update);
            } else {
                overflow = true;
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            int j = j_start + i + totThreads*ii;
            if (j < tsize) {
                m[j] = mi[ii];
                v[j] = vi[ii];
                p[j] = pi[ii];
                if (p_copy != NULL) p_copy[j] = static_cast<GRAD_T>(pi[ii]);
            }
        }
    }

    if (p_copy != NULL) {
        __syncthreads();
        if (overflow) {
            p_copy[0] = INFINITY;
        }
    }
}

template <typename T, typename GRAD_T>
__global__ void adam_undo_cuda_kernel(
        T* __restrict__ p,
        T* __restrict__ m,
        T* __restrict__ v,
        const GRAD_T * __restrict__ g,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        const size_t tsize,
        adamMode_t mode,
        const float decay)
{
    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock);
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock;

    T mi[ILP];
    T vi[ILP];
    T pi[ILP];
    T gi[ILP];

    for(int j_start = 0;  j_start < tsize;  j_start+=totThreads*ILP) {
#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            mi[ii] = T(0);
            vi[ii] = T(0);
            pi[ii] = T(0);
            gi[ii] = GRAD_T(0);

            int j = j_start + i*ILP;
            if (j < tsize) {
                pi[ii] = p[j];
                mi[ii] = m[j];
                vi[ii] = v[j];
                gi[ii] = static_cast<T>(g[j]);
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            T scaled_grad = gi[ii]/grad_scale;
            if (isfinite(scaled_grad)) {
                float denom;
                if (mode == ADAM_MODE_0)
                    denom = sqrtf(vi[ii] + eps);
                else // Mode 1
                    denom = sqrtf(vi[ii]) + eps;
                pi[ii] = (pi[ii] + step_size*(mi[ii]/denom)) / (1.0f - step_size*decay);
                mi[ii] = (mi[ii] - (1-b1)*scaled_grad) / b1;
                vi[ii] = (vi[ii] - (1-b2)*scaled_grad*scaled_grad) / b2;
                // Make sure round off errors don't create (small) negative value.
                // This can happen if we have to revert the very first step.
                vi[ii] = vi[ii] >= 0.0f ? vi[ii] : 0.0f;
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            int j = j_start + i*ILP;
            if (j < tsize) {
                m[j] = mi[ii];
                v[j] = vi[ii];
                p[j] = pi[ii];
            }
        }
    }
}

template <int DEPTH, typename T, typename GRAD_T>
struct AdamFunctor
{
    __device__ __forceinline__ void operator()(
        int chunk_size,
        volatile int* noop_gmem,
        TensorListMetadata<DEPTH>& tl,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        adamMode_t mode,
        const float decay)
    {
        int tensor_loc = tl.block_to_tensor[blockIdx.x];
        int chunk_idx = tl.block_to_chunk[blockIdx.x];
        int n = tl.sizes[tensor_loc];

        T* p = (T *)tl.addresses[0][tensor_loc];
        p += chunk_idx*chunk_size;
        T* m = (T *)tl.addresses[1][tensor_loc];
        m += chunk_idx*chunk_size;
        T* v = (T *)tl.addresses[2][tensor_loc];
        v += chunk_idx*chunk_size;
        GRAD_T* g = (GRAD_T *)tl.addresses[3][tensor_loc];
        g += chunk_idx*chunk_size;
        GRAD_T* p_copy = NULL;
        if (DEPTH == 5) {
            p_copy = (GRAD_T *)tl.addresses[4][tensor_loc];
            p_copy += chunk_idx*chunk_size;
        }

        n -= chunk_idx*chunk_size;
        int dim = chunk_size < n ? chunk_size : n;

        T mi[ILP];
        T vi[ILP];
        T pi[ILP];
        T gi[ILP];

        bool overflow = false;
        for(int j_start = 0;  j_start < dim;  j_start+=blockDim.x*ILP) {
#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                mi[ii] = T(0);
                vi[ii] = T(0);
                pi[ii] = T(0);
                gi[ii] = GRAD_T(0);

                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    pi[ii] = p[j];
                    mi[ii] = m[j];
                    vi[ii] = v[j];
                    gi[ii] = static_cast<T>(g[j]);
                }
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                T scaled_grad = gi[ii]/grad_scale;
                if (isfinite(scaled_grad)) {
                    mi[ii] = b1*mi[ii] + (1-b1)*scaled_grad;
                    vi[ii] = b2*vi[ii] + (1-b2)*scaled_grad*scaled_grad;
                    float denom;
                    if (mode == ADAM_MODE_0)
                        denom = sqrtf(vi[ii] + eps);
                    else // Mode 1
                        denom = sqrtf(vi[ii]) + eps;
                    float update = (mi[ii]/denom) + (decay*pi[ii]);
                    pi[ii] = pi[ii] - (step_size*update);
                } else {
                    overflow = true;
                }
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    m[j] = mi[ii];
                    v[j] = vi[ii];
                    p[j] = pi[ii];
                    if (p_copy != NULL) p_copy[j] = static_cast<GRAD_T>(pi[ii]);
                }
            }
        }

        if (overflow) {
            *noop_gmem = 1;
        }
    }
};

template <int DEPTH, typename T, typename GRAD_T>
struct AdamUndoFunctor
{
    __device__ __forceinline__ void operator()(
        int chunk_size,
        volatile int* noop_gmem,
        TensorListMetadata<DEPTH>& tl,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        adamMode_t mode,
        const float decay)
    {
        int tensor_loc = tl.block_to_tensor[blockIdx.x];
        int chunk_idx = tl.block_to_chunk[blockIdx.x];
        int n = tl.sizes[tensor_loc];

        T* p = (T *)tl.addresses[0][tensor_loc];
        p += chunk_idx*chunk_size;
        T* m = (T *)tl.addresses[1][tensor_loc];
        m += chunk_idx*chunk_size;
        T* v = (T *)tl.addresses[2][tensor_loc];
        v += chunk_idx*chunk_size;
        GRAD_T* g = (GRAD_T *)tl.addresses[3][tensor_loc];
        g += chunk_idx*chunk_size;

        n -= chunk_idx*chunk_size;
        int dim = chunk_size < n ? chunk_size : n;

        T mi[ILP];
        T vi[ILP];
        T pi[ILP];
        T gi[ILP];

        for(int j_start = 0;  j_start < dim;  j_start+=blockDim.x*ILP) {
#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                mi[ii] = T(0);
                vi[ii] = T(0);
                pi[ii] = T(0);
                gi[ii] = GRAD_T(0);

                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    pi[ii] = p[j];
                    mi[ii] = m[j];
                    vi[ii] = v[j];
                    gi[ii] = static_cast<T>(g[j]);
                }
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                T scaled_grad = gi[ii]/grad_scale;
                if (isfinite(scaled_grad)) {
                    float denom;
                    if (mode == ADAM_MODE_0)
                        denom = sqrtf(vi[ii] + eps);
                    else // Mode 1
                        denom = sqrtf(vi[ii]) + eps;
                    pi[ii] = (pi[ii] + step_size*(mi[ii]/denom)) / (1.0f - step_size*decay);
                    mi[ii] = (mi[ii] - (1-b1)*scaled_grad) / b1;
                    vi[ii] = (vi[ii] - (1-b2)*scaled_grad*scaled_grad) / b2;
                    // Make sure round off errors don't create (small) negative value.
                    // This can happen if we have to revert the very first step.
                    vi[ii] = vi[ii] >= 0.0f ? vi[ii] : 0.0f;
                }
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    m[j] = mi[ii];
                    v[j] = vi[ii];
                    p[j] = pi[ii];
                }
            }
        }
    }
};

void fused_strided_check_finite(
	at::Tensor & noop,
        at::Tensor & p_copy,
        int stride,
	int clear_overflow_first)
{
	//Get tensor size
	int tsize = p_copy.numel();
	int niter = (tsize + stride - 1) / stride;

	//Determine #threads and #blocks
	const int threadsPerBlock = 512;
	const dim3 blocks((niter+threadsPerBlock-1)/threadsPerBlock);
	AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p_copy), "parameter tensor is too large to be indexed with int32");

	hipStream_t stream = at::cuda::getCurrentCUDAStream();
	using namespace at; // prevents "toString is undefined" errors
	DISPATCH_FLOAT_AND_HALF(p_copy.scalar_type(), 0, "check_finite_cuda_kernel",
			strided_check_finite_cuda_kernel<scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
				noop.DATA_PTR<int>(),
				p_copy.DATA_PTR<scalar_t_0>(),
				tsize,
				stride,
				clear_overflow_first);
			);
	THCudaCheck(hipGetLastError());
}

void fused_adam_cuda(
        at::Tensor & p,
        at::Tensor & p_copy,
        at::Tensor & m,
        at::Tensor & v,
        at::Tensor & g,
        float lr,
        float beta1,
        float beta2,
        float eps,
        float grad_scale,
        int step,
        int mode,
        int bias_correction,
        float decay)
{
//        using namespace at;

        //Get tensor size
        int tsize = p.numel();
        //Determine #threads and #blocks
        const int threadsPerBlock = 512;
        const dim3 blocks((tsize+threadsPerBlock-1)/threadsPerBlock);
        AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p), "parameter tensor is too large to be indexed with int32");
        //Constants
        float step_size = 0;
        if (bias_correction == 1) {
            const float bias_correction1 = 1 - std::pow(beta1, step);
            const float bias_correction2 = 1 - std::pow(beta2, step);
            step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
        }
        else {
            step_size = lr;
        }
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        if (g.scalar_type() == at::ScalarType::Half) {
//all other values should be fp32 for half gradients
            AT_ASSERTM(p.scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
//dispatch is done on the gradient type
            using namespace at; // prevents "toString is undefined" errors
            DISPATCH_FLOAT_AND_HALF(g.scalar_type(), 0, "adam_cuda_kernel",
                using accscalar_t = at::acc_type<scalar_t_0, true>;
                adam_cuda_kernel<accscalar_t, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                        p.DATA_PTR<accscalar_t>(),
                        p_copy.numel() ? p_copy.DATA_PTR<scalar_t_0>() : NULL,
                        m.DATA_PTR<accscalar_t>(),
                        v.DATA_PTR<accscalar_t>(),
                        g.DATA_PTR<scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        tsize,
                        (adamMode_t) mode,
                        decay);
                );
      } else {
            using namespace at;
            DISPATCH_DOUBLE_AND_FLOAT(g.scalar_type(), 0, "adam_cuda_kernel",
                adam_cuda_kernel<scalar_t_0, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                        p.DATA_PTR<scalar_t_0>(),
                        NULL, //don't output p_copy for fp32, it's wasted write
                        m.DATA_PTR<scalar_t_0>(),
                        v.DATA_PTR<scalar_t_0>(),
                        g.DATA_PTR<scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        tsize,
                        (adamMode_t) mode,
                        decay);
            );
      }
      THCudaCheck(hipGetLastError());

}

void fused_adam_undo_cuda(
        at::Tensor & p,
        at::Tensor & m,
        at::Tensor & v,
        at::Tensor & g,
        float lr,
        float beta1,
        float beta2,
        float eps,
        float grad_scale,
        int step,
        int mode,
        int bias_correction,
        float decay)
{
//        using namespace at;

        //Get tensor size
        int tsize = p.numel();
        //Determine #threads and #blocks
        const int threadsPerBlock = 512;
        const dim3 blocks((tsize+threadsPerBlock-1)/threadsPerBlock);
        AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p), "parameter tensor is too large to be indexed with int32");
        //Constants
        float step_size = 0;
        if (bias_correction == 1) {
            const float bias_correction1 = 1 - std::pow(beta1, step);
            const float bias_correction2 = 1 - std::pow(beta2, step);
            step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
        }
        else {
            step_size = lr;
        }
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        if (g.scalar_type() == at::ScalarType::Half) {
//all other values should be fp32 for half gradients
            AT_ASSERTM(p.scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
//dispatch is done on the gradient type
            using namespace at; // prevents "toString is undefined" errors
            DISPATCH_FLOAT_AND_HALF(g.scalar_type(), 0, "adam_cuda_kernel",
                using accscalar_t = at::acc_type<scalar_t_0, true>;
                adam_undo_cuda_kernel<accscalar_t, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                        p.DATA_PTR<accscalar_t>(),
                        m.DATA_PTR<accscalar_t>(),
                        v.DATA_PTR<accscalar_t>(),
                        g.DATA_PTR<scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        tsize,
                        (adamMode_t) mode,
                        decay);
                );
      } else {
            using namespace at;
            DISPATCH_DOUBLE_AND_FLOAT(g.scalar_type(), 0, "adam_cuda_kernel",
                adam_undo_cuda_kernel<scalar_t_0, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                        p.DATA_PTR<scalar_t_0>(),
                        m.DATA_PTR<scalar_t_0>(),
                        v.DATA_PTR<scalar_t_0>(),
                        g.DATA_PTR<scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        tsize,
                        (adamMode_t) mode,
                        decay);
            );
      }
      THCudaCheck(hipGetLastError());

}

void fused_adam_cuda_mt(
    int chunk_size,
    at::Tensor noop_flag,
    std::vector<std::vector<at::Tensor>> tensor_lists, // p, m, v, g, p_copy
    float lr,
    float beta1,
    float beta2,
    float eps,
    float grad_scale,
    int step,
    int mode,
    int bias_correction,
    float decay) {

    //Constants
    float step_size = 0;
    if (bias_correction == 1) {
        const float bias_correction1 = 1 - std::pow(beta1, step);
        const float bias_correction2 = 1 - std::pow(beta2, step);
        step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
    }
    else {
        step_size = lr;
    }
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    size_t tl_sz = tensor_lists.size();
    AT_ASSERTM(tl_sz == 4 || tl_sz == 5, "expected tensor lists of size 4 or 5");

    if (tensor_lists[3][0].scalar_type() == at::ScalarType::Half) {
//alher values should be fp32 for half gradients
        AT_ASSERTM(tensor_lists[0][0].scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
//dich is done on the gradient type
        if (tl_sz == 5) {
            DISPATCH_FLOAT_AND_HALF(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                    using accscalar_t = at::acc_type<scalar_t_0, true>;
                    multi_tensor_apply<5>(
                        BLOCK_SIZE,
                        chunk_size,
                        noop_flag,
                        tensor_lists,
                        AdamFunctor<5, accscalar_t, scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        (adamMode_t) mode,
                        decay);
                    );
        } else {
            DISPATCH_FLOAT_AND_HALF(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                    using accscalar_t = at::acc_type<scalar_t_0, true>;
                    multi_tensor_apply<4>(
                        BLOCK_SIZE,
                        chunk_size,
                        noop_flag,
                        tensor_lists,
                        AdamFunctor<4, accscalar_t, scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        (adamMode_t) mode,
                        decay);
                    );
        }
    } else {
        if (tl_sz == 5) {
            DISPATCH_DOUBLE_AND_FLOAT(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                    multi_tensor_apply<5>(
                        BLOCK_SIZE,
                        chunk_size,
                        noop_flag,
                        tensor_lists,
                        AdamFunctor<5, scalar_t_0, scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        (adamMode_t) mode,
                        decay);
                    );
        } else {
            DISPATCH_DOUBLE_AND_FLOAT(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                    multi_tensor_apply<4>(
                        BLOCK_SIZE,
                        chunk_size,
                        noop_flag,
                        tensor_lists,
                        AdamFunctor<4, scalar_t_0, scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        (adamMode_t) mode,
                        decay);
                    );
        }
    }
    THCudaCheck(hipGetLastError());
}

void fused_adam_undo_cuda_mt(
    int chunk_size,
    at::Tensor noop_flag,
    std::vector<std::vector<at::Tensor>> tensor_lists, // p, m, v, g, p_copy
    float lr,
    float beta1,
    float beta2,
    float eps,
    float grad_scale,
    int step,
    int mode,
    int bias_correction,
    float decay) {

    //Constants
    float step_size = 0;
    if (bias_correction == 1) {
        const float bias_correction1 = 1 - std::pow(beta1, step);
        const float bias_correction2 = 1 - std::pow(beta2, step);
        step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
    }
    else {
        step_size = lr;
    }
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    size_t tl_sz = tensor_lists.size();
    AT_ASSERTM(tl_sz == 4, "expected tensor list of size 4");

    if (tensor_lists[3][0].scalar_type() == at::ScalarType::Half) {
        //alher values should be fp32 for half gradients
        AT_ASSERTM(tensor_lists[0][0].scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
        //dich is done on the gradient type
        DISPATCH_FLOAT_AND_HALF(tensor_lists[3][0].scalar_type(), 0, "adam_undo_cuda_mt_kernel",
                using accscalar_t = at::acc_type<scalar_t_0, true>;
                multi_tensor_apply<4>(
                    BLOCK_SIZE,
                    chunk_size,
                    noop_flag,
                    tensor_lists,
                    AdamUndoFunctor<4, accscalar_t, scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    (adamMode_t) mode,
                    decay);
                );
    } else {
        DISPATCH_DOUBLE_AND_FLOAT(tensor_lists[3][0].scalar_type(), 0, "adam_undo_cuda_mt_kernel",
                multi_tensor_apply<4>(
                    BLOCK_SIZE,
                    chunk_size,
                    noop_flag,
                    tensor_lists,
                    AdamUndoFunctor<4, scalar_t_0, scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    (adamMode_t) mode,
                    decay);
                );
    }
    THCudaCheck(hipGetLastError());
}

