#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/cuda/detail/IndexUtils.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include "ATen/TensorUtils.h"
// #include "ATen/Type.h"
#include "ATen/AccumulateType.h"
#include <THC/THCGeneral.h>
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

#include "type_shim.h"

typedef enum{
    ADAM_MODE_0   =0, // eps under square root
    ADAM_MODE_1   =1  // eps outside square root
} adamMode_t;

template <typename FROM_T, typename TO_T> 
__device__ void convert(const FROM_T vi, TO_T& vo)
{
    vo = static_cast<TO_T>(vi);
}

template <>
__device__ void convert(const float vi, uint8_t& vo)
{
    union S
    {
	float as_float;
	int as_int;
    };
    S s;
    s.as_float = vi;
    s.as_int = s.as_int & 0xFF800000;
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_half = static_cast<at::Half>(vi + s.as_float / 8.0f);
    vo = t.as_byte[1];
}

template <>
__device__ void convert(const uint8_t vi, float& vo)
{
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_byte[0] = 0;
    t.as_byte[1] = vi;
    vo = static_cast<float>(t.as_half);
}

template <>
__device__ void convert(const at::Half vi, uint8_t& vo)
{
    union S
    {
	float as_float;
	int as_int;
    };
    S s;
    s.as_float = static_cast<float>(vi);
    s.as_int = s.as_int & 0xFF800000;
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_half = static_cast<at::Half>(vi + s.as_float / 8.0f);
    vo = t.as_byte[1];
}

template <>
__device__ void convert(const uint8_t vi, at::Half& vo)
{
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_byte[0] = 0;
    t.as_byte[1] = vi;
    vo = t.as_half;
}

template <typename GRAD_T>
__global__ void strided_check_finite_cuda_kernel(
        volatile int* noop_gmem,
        GRAD_T* __restrict__ p_copy,
        const size_t tsize,
        int stride,
        int clear_overflow_first)
{
    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock) * stride;
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock*stride;

    if (clear_overflow_first) {
        if (i == 0) {
            *noop_gmem = 0;
        }
        __syncthreads();
    }

    for (int j = i; j < tsize; j+=totThreads) {
        GRAD_T pi = p_copy[j];
        if (!isfinite(pi)) {
            *noop_gmem = 1;
        }
    }
}
template <>
__global__ void strided_check_finite_cuda_kernel(
        volatile int* noop_gmem,
        uint8_t* __restrict__ p_copy,
        const size_t tsize,
        int stride,
        int clear_overflow_first)
{
    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock) * stride;
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock*stride;

    if (clear_overflow_first) {
        if (i == 0) {
            *noop_gmem = 0;
        }
        __syncthreads();
    }

    for (int j = i; j < tsize; j+=totThreads) {
        at::Half pi;
        convert(p_copy[j], pi);
        if (!isfinite(pi)) {
            *noop_gmem = 1;
        }
    }
}

template <typename FROM_T, typename TO_T> 
__global__ void maybe_cast_kernel(
        volatile int* overflow_flag,
        const FROM_T* p_in,
        TO_T* p_out,
        const size_t tsize)
{
    if (overflow_flag && *overflow_flag != 0) return;

    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock);
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock;

    FROM_T pi[ILP];
    TO_T po[ILP];

    for(int j_start = 0;  j_start < tsize;  j_start+=totThreads*ILP) {
#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            pi[ii] = 0;

            int j = j_start + i + totThreads*ii;
            if (j < tsize) {
                pi[ii] = p_in[j];
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            convert(pi[ii], po[ii]);
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            int j = j_start + i + totThreads*ii;
            if (j < tsize) {
                p_out[j] = po[ii];
            }
        }
    }
}

template <typename T, typename GRAD_T, typename REDU_T>
__global__ void adam_cuda_kernel(
        T* __restrict__ p,
        REDU_T* __restrict__ p_copy, // For mixed precision training, pass NULL if not needed
        T* __restrict__ m,
        T* __restrict__ v,
        const GRAD_T * __restrict__ g,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        const size_t tsize,
        adamMode_t mode,
        const float decay)
{
    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock);
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock;

    T mi[ILP];
    T vi[ILP];
    T pi[ILP];
    T gi[ILP];

    bool overflow = false;
    for(int j_start = 0;  j_start < tsize;  j_start+=totThreads*ILP) {
#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            mi[ii] = T(0);
            vi[ii] = T(0);
            pi[ii] = T(0);
            gi[ii] = GRAD_T(0);

            int j = j_start + i + totThreads*ii;
            if (j < tsize) {
                pi[ii] = p[j];
                mi[ii] = m[j];
                vi[ii] = v[j];
                gi[ii] = static_cast<T>(g[j]);
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            T scaled_grad = gi[ii]/grad_scale;
            if (isfinite(scaled_grad)) {
                mi[ii] = b1*mi[ii] + (1-b1)*scaled_grad;
                vi[ii] = b2*vi[ii] + (1-b2)*scaled_grad*scaled_grad;
                float denom;
                if (mode == ADAM_MODE_0)
                    denom = sqrtf(vi[ii] + eps);
                else // Mode 1
                    denom = sqrtf(vi[ii]) + eps;
                float update = (mi[ii]/denom) + (decay*pi[ii]);
                pi[ii] = pi[ii] - (step_size*update);
            } else {
                overflow = true;
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            int j = j_start + i + totThreads*ii;
            if (j < tsize) {
                m[j] = mi[ii];
                v[j] = vi[ii];
                p[j] = pi[ii];
                if (p_copy != NULL) {
                    convert(pi[ii], p_copy[j]);
                }
            }
        }
    }

    if (p_copy != NULL) {
        __syncthreads();
        if (overflow) {
            convert(float(INFINITY), p_copy[0]);
        }
    }
}

template <typename T, typename GRAD_T>
__global__ void maybe_adam_undo_cuda_kernel(
        volatile int* overflow_flag,
        T* __restrict__ p,
        T* __restrict__ m,
        T* __restrict__ v,
        const GRAD_T * __restrict__ g,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        const size_t tsize,
        adamMode_t mode,
        const float decay)
{
    // NB! Skip undo kernel when overflow flag is NOT set
    if (overflow_flag && *overflow_flag == 0) return;

    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock);
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock;

    T mi[ILP];
    T vi[ILP];
    T pi[ILP];
    T gi[ILP];

    for(int j_start = 0;  j_start < tsize;  j_start+=totThreads*ILP) {
#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            mi[ii] = T(0);
            vi[ii] = T(0);
            pi[ii] = T(0);
            gi[ii] = GRAD_T(0);

            int j = j_start + i*ILP;
            if (j < tsize) {
                pi[ii] = p[j];
                mi[ii] = m[j];
                vi[ii] = v[j];
                gi[ii] = static_cast<T>(g[j]);
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            T scaled_grad = gi[ii]/grad_scale;
            if (isfinite(scaled_grad)) {
                float denom;
                if (mode == ADAM_MODE_0)
                    denom = sqrtf(vi[ii] + eps);
                else // Mode 1
                    denom = sqrtf(vi[ii]) + eps;
                pi[ii] = (pi[ii] + step_size*(mi[ii]/denom)) / (1.0f - step_size*decay);
                mi[ii] = (mi[ii] - (1-b1)*scaled_grad) / b1;
                vi[ii] = (vi[ii] - (1-b2)*scaled_grad*scaled_grad) / b2;
                // Make sure round off errors don't create (small) negative value.
                // This can happen if we have to revert the very first step.
                vi[ii] = vi[ii] >= 0.0f ? vi[ii] : 0.0f;
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            int j = j_start + i*ILP;
            if (j < tsize) {
                m[j] = mi[ii];
                v[j] = vi[ii];
                p[j] = pi[ii];
            }
        }
    }
}

__global__ void update_step_and_loss_scaler_kernel(
        volatile int* overflow_flag,
	double* __restrict__ step_and_loss_scaler_vec)
{
    // 0 : step
    // 1 : iter
    // 2 : loss_scale
    // 3 : last_overflow_iter
    // 4 : scale_factor
    // 5 : scale_window
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0) {
        double loss_scale = step_and_loss_scaler_vec[2];
        double scale_factor = step_and_loss_scaler_vec[4];
        int iter = static_cast<int>(step_and_loss_scaler_vec[1]);
        int last_overflow_iter = static_cast<int>(step_and_loss_scaler_vec[3]);
        if (*overflow_flag == 0) {
            // increase step
            step_and_loss_scaler_vec[0] += 1.0;
            // maybe increase loss scaler
            int scale_window = static_cast<int>(step_and_loss_scaler_vec[5]);
            if (((iter - last_overflow_iter) % scale_window) == 0) {
                step_and_loss_scaler_vec[2] = loss_scale * scale_factor;
            }
        } else {
            step_and_loss_scaler_vec[2] = loss_scale / scale_factor;
            step_and_loss_scaler_vec[3] = static_cast<double>(iter);
        }
        step_and_loss_scaler_vec[1] += 1.0;
    }
}

template <int DEPTH, typename FROM_T, typename TO_T>
struct MaybeCastFunctor
{
    __device__ __forceinline__ void operator()(
        int chunk_size,
        volatile int* overflow_flag,
        TensorListMetadata<DEPTH>& tl)
    {
        if (overflow_flag && *overflow_flag != 0) return;

        int tensor_loc = tl.block_to_tensor[blockIdx.x];
        int chunk_idx = tl.block_to_chunk[blockIdx.x];
        int n = tl.sizes[tensor_loc];

        FROM_T* p_in = (FROM_T *)tl.addresses[0][tensor_loc];
        p_in += chunk_idx*chunk_size;
        TO_T* p_out = (TO_T *)tl.addresses[1][tensor_loc];
        p_out += chunk_idx*chunk_size;

        n -= chunk_idx*chunk_size;
        int dim = chunk_size < n ? chunk_size : n;

	FROM_T pi[ILP];
        TO_T po[ILP];

        for(int j_start = 0;  j_start < dim;  j_start+=blockDim.x*ILP) {
#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                pi[ii] = FROM_T(0);
                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    pi[ii] = p_in[j];
                }
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                convert(pi[ii], po[ii]);
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    p_out[j] = po[ii];
                }
            }
        }
    }
};

template <int DEPTH, typename T, typename GRAD_T>
struct AdamFunctor
{
    __device__ __forceinline__ void operator()(
        int chunk_size,
        volatile int* overflow_flag,
        TensorListMetadata<DEPTH>& tl,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        adamMode_t mode,
        const float decay)
    {
        int tensor_loc = tl.block_to_tensor[blockIdx.x];
        int chunk_idx = tl.block_to_chunk[blockIdx.x];
        int n = tl.sizes[tensor_loc];

        T* p = (T *)tl.addresses[0][tensor_loc];
        p += chunk_idx*chunk_size;
        T* m = (T *)tl.addresses[1][tensor_loc];
        m += chunk_idx*chunk_size;
        T* v = (T *)tl.addresses[2][tensor_loc];
        v += chunk_idx*chunk_size;
        GRAD_T* g = (GRAD_T *)tl.addresses[3][tensor_loc];
        g += chunk_idx*chunk_size;
        GRAD_T* p_copy = NULL;
        if (DEPTH == 5) {
            p_copy = (GRAD_T *)tl.addresses[4][tensor_loc];
            p_copy += chunk_idx*chunk_size;
        }

        n -= chunk_idx*chunk_size;
        int dim = chunk_size < n ? chunk_size : n;

        T mi[ILP];
        T vi[ILP];
        T pi[ILP];
        T gi[ILP];

        bool overflow = false;
        for(int j_start = 0;  j_start < dim;  j_start+=blockDim.x*ILP) {
#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                mi[ii] = T(0);
                vi[ii] = T(0);
                pi[ii] = T(0);
                gi[ii] = GRAD_T(0);

                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    pi[ii] = p[j];
                    mi[ii] = m[j];
                    vi[ii] = v[j];
                    gi[ii] = static_cast<T>(g[j]);
                }
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                T scaled_grad = gi[ii]/grad_scale;
                if (isfinite(scaled_grad)) {
                    mi[ii] = b1*mi[ii] + (1-b1)*scaled_grad;
                    vi[ii] = b2*vi[ii] + (1-b2)*scaled_grad*scaled_grad;
                    float denom;
                    if (mode == ADAM_MODE_0)
                        denom = sqrtf(vi[ii] + eps);
                    else // Mode 1
                        denom = sqrtf(vi[ii]) + eps;
                    float update = (mi[ii]/denom) + (decay*pi[ii]);
                    pi[ii] = pi[ii] - (step_size*update);
                } else {
                    overflow = true;
                }
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    m[j] = mi[ii];
                    v[j] = vi[ii];
                    p[j] = pi[ii];
                    if (p_copy != NULL) p_copy[j] = static_cast<GRAD_T>(pi[ii]);
                }
            }
        }

        if (overflow) {
            *overflow_flag = 1;
        }
    }
};

template <int DEPTH, typename T, typename GRAD_T>
struct MaybeAdamUndoFunctor
{
    __device__ __forceinline__ void operator()(
        int chunk_size,
        volatile int* overflow_flag,
        TensorListMetadata<DEPTH>& tl,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        adamMode_t mode,
        const float decay)
    {
        // Skip Adam undo when overflow flag is NOT set
        if (overflow_flag && *overflow_flag == 0) return;

        int tensor_loc = tl.block_to_tensor[blockIdx.x];
        int chunk_idx = tl.block_to_chunk[blockIdx.x];
        int n = tl.sizes[tensor_loc];

        T* p = (T *)tl.addresses[0][tensor_loc];
        p += chunk_idx*chunk_size;
        T* m = (T *)tl.addresses[1][tensor_loc];
        m += chunk_idx*chunk_size;
        T* v = (T *)tl.addresses[2][tensor_loc];
        v += chunk_idx*chunk_size;
        GRAD_T* g = (GRAD_T *)tl.addresses[3][tensor_loc];
        g += chunk_idx*chunk_size;

        n -= chunk_idx*chunk_size;
        int dim = chunk_size < n ? chunk_size : n;

        T mi[ILP];
        T vi[ILP];
        T pi[ILP];
        T gi[ILP];

        for(int j_start = 0;  j_start < dim;  j_start+=blockDim.x*ILP) {
#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                mi[ii] = T(0);
                vi[ii] = T(0);
                pi[ii] = T(0);
                gi[ii] = GRAD_T(0);

                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    pi[ii] = p[j];
                    mi[ii] = m[j];
                    vi[ii] = v[j];
                    gi[ii] = static_cast<T>(g[j]);
                }
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                T scaled_grad = gi[ii]/grad_scale;
                if (isfinite(scaled_grad)) {
                    float denom;
                    if (mode == ADAM_MODE_0)
                        denom = sqrtf(vi[ii] + eps);
                    else // Mode 1
                        denom = sqrtf(vi[ii]) + eps;
                    pi[ii] = (pi[ii] + step_size*(mi[ii]/denom)) / (1.0f - step_size*decay);
                    mi[ii] = (mi[ii] - (1-b1)*scaled_grad) / b1;
                    vi[ii] = (vi[ii] - (1-b2)*scaled_grad*scaled_grad) / b2;
                    // Make sure round off errors don't create (small) negative value.
                    // This can happen if we have to revert the very first step.
                    vi[ii] = vi[ii] >= 0.0f ? vi[ii] : 0.0f;
                }
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    m[j] = mi[ii];
                    v[j] = vi[ii];
                    p[j] = pi[ii];
                }
            }
        }
    }
};

void fused_strided_check_finite(
	at::Tensor & overflow_flag,
        at::Tensor & p_copy,
        int stride,
	int clear_overflow_first)
{
	//Get tensor size
	int tsize = p_copy.numel();
	int niter = (tsize + stride - 1) / stride;

	//Determine #threads and #blocks
	const int threadsPerBlock = 512;
	const dim3 blocks((niter+threadsPerBlock-1)/threadsPerBlock);
	AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p_copy), "parameter tensor is too large to be indexed with int32");

	hipStream_t stream = at::cuda::getCurrentCUDAStream();
        using namespace at; // prevents "toString is undefined" errors
        DISPATCH_FLOAT_HALF_AND_BYTE(p_copy.scalar_type(), 0, "check_finite_cuda_kernel",
                strided_check_finite_cuda_kernel<scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                    overflow_flag.DATA_PTR<int>(),
                    p_copy.DATA_PTR<scalar_t_0>(),
                    tsize,
                    stride,
                    clear_overflow_first);
                );
	THCudaCheck(hipGetLastError());
}

void fused_adam_cuda(
        at::Tensor & p,
        at::Tensor & p_copy,
        at::Tensor & m,
        at::Tensor & v,
        at::Tensor & g,
        float lr,
        float beta1,
        float beta2,
        float eps,
        float grad_scale,
        int step,
        int mode,
        int bias_correction,
        float decay)
{
//      using namespace at;

      //Get tensor size
      int tsize = p.numel();
      //Determine #threads and #blocks
      const int threadsPerBlock = 512;
      const dim3 blocks((tsize+threadsPerBlock-1)/threadsPerBlock);
      AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p), "parameter tensor is too large to be indexed with int32");
      //Constants
      float step_size = 0;
      if (bias_correction == 1) {
          const float bias_correction1 = 1 - std::pow(beta1, step);
          const float bias_correction2 = 1 - std::pow(beta2, step);
          step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
      }
      else {
          step_size = lr;
      }
      hipStream_t stream = at::cuda::getCurrentCUDAStream();

      if (g.scalar_type() == at::ScalarType::Half) {
          //all other values should be fp32 for half gradients
          AT_ASSERTM(p.scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
          //dispatch is done on the gradient type
          using namespace at; // prevents "toString is undefined" errors
          if (p_copy.numel() == 0 || p_copy.scalar_type() == g.scalar_type()) {
              DISPATCH_FLOAT_AND_HALF(g.scalar_type(), 0, "adam_cuda_kernel",
                      using accscalar_t = at::acc_type<scalar_t_0, true>;
                      adam_cuda_kernel<accscalar_t, scalar_t_0, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                          p.DATA_PTR<accscalar_t>(),
                          p_copy.numel() ? p_copy.DATA_PTR<scalar_t_0>() : NULL,
                          m.DATA_PTR<accscalar_t>(),
                          v.DATA_PTR<accscalar_t>(),
                          g.DATA_PTR<scalar_t_0>(),
                          beta1,
                          beta2,
                          eps,
                          grad_scale,
                          step_size,
                          tsize,
                          (adamMode_t) mode,
                          decay);
                      );
          } else {
              AT_ASSERTM(p_copy.scalar_type() == at::ScalarType::Byte, "expected parameter to be of byte type");
              DISPATCH_FLOAT_AND_HALF(g.scalar_type(), 0, "adam_cuda_e5m2_kernel",
                      using accscalar_t = at::acc_type<scalar_t_0, true>;
                      adam_cuda_kernel<accscalar_t, scalar_t_0, uint8_t><<<blocks,threadsPerBlock, 0, stream>>>(
                          p.DATA_PTR<accscalar_t>(),
                          p_copy.DATA_PTR<uint8_t>(),
                          m.DATA_PTR<accscalar_t>(),
                          v.DATA_PTR<accscalar_t>(),
                          g.DATA_PTR<scalar_t_0>(),
                          beta1,
                          beta2,
                          eps,
                          grad_scale,
                          step_size,
                          tsize,
                          (adamMode_t) mode,
                          decay);
                      );
          }
      } else {
          using namespace at;
          DISPATCH_DOUBLE_AND_FLOAT(g.scalar_type(), 0, "adam_cuda_kernel",
                  adam_cuda_kernel<scalar_t_0, scalar_t_0, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                      p.DATA_PTR<scalar_t_0>(),
                      NULL, //don't output p_copy for fp32, it's wasted write
                      m.DATA_PTR<scalar_t_0>(),
                      v.DATA_PTR<scalar_t_0>(),
                      g.DATA_PTR<scalar_t_0>(),
                      beta1,
                      beta2,
                      eps,
                      grad_scale,
                      step_size,
                      tsize,
                      (adamMode_t) mode,
                      decay);
                  );
      }
      THCudaCheck(hipGetLastError());
}

void maybe_cast_cuda(
        at::Tensor & overflow_flag,
        at::Tensor & p_in,
        at::Tensor & p_out)
{
      //Get tensor size
      int tsize = p_in.numel();
      AT_ASSERTM(tsize == p_out.numel(), "p_in.numel() must equal p_out.numel()");
      //Determine #threads and #blocks
      const int threadsPerBlock = 512;
      const dim3 blocks((tsize+threadsPerBlock-1)/threadsPerBlock);
      AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p_in), "parameter tensor is too large to be indexed with int32");
      //Constants
      hipStream_t stream = at::cuda::getCurrentCUDAStream();
      DISPATCH_FLOAT_HALF_AND_BYTE(p_in.scalar_type(), 0, "maybe_cast_cuda"
              DISPATCH_FLOAT_HALF_AND_BYTE(p_out.scalar_type(), 1, "maybe_cast_cuda",
                  maybe_cast_kernel<scalar_t_0,scalar_t_1><<<blocks,threadsPerBlock, 0, stream>>>(
                      overflow_flag.numel() ? overflow_flag.DATA_PTR<int>() : NULL,
                      p_in.DATA_PTR<scalar_t_0>(),
                      p_out.DATA_PTR<scalar_t_1>(),
                      tsize); ))
      THCudaCheck(hipGetLastError());
}

void maybe_cast_cuda_mt(
    int chunk_size,
    at::Tensor overflow_flag,
    std::vector<std::vector<at::Tensor>> tensor_lists) // p_in, p_out
{
    //Constants
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    size_t tl_sz = tensor_lists.size();
    AT_ASSERTM(tl_sz == 2, "expected tensor lists of size 2");

    DISPATCH_FLOAT_HALF_AND_BYTE(tensor_lists[0][0].scalar_type(), 0, "maybe_cast_cuda_mt_kernel",
            DISPATCH_FLOAT_HALF_AND_BYTE(tensor_lists[1][0].scalar_type(), 1, "maybe_cast_cuda_mt_kernel",
                multi_tensor_apply<2>(
                    BLOCK_SIZE,
                    chunk_size,
                    overflow_flag,
                    tensor_lists,
                    MaybeCastFunctor<2, scalar_t_0, scalar_t_1>()); ))
    THCudaCheck(hipGetLastError());
}

void update_step_and_loss_scaler_cuda(
        at::Tensor & overflow_flag,
        at::Tensor & step_and_loss_scaler)
{
    AT_ASSERTM(step_and_loss_scaler.numel() == 6, "step_and_loss_scaler must have 6 elements");
    AT_ASSERTM(step_and_loss_scaler.scalar_type() == at::ScalarType::Double, "expected step_and_loss_scaler to be a double tensor");
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    update_step_and_loss_scaler_kernel<<<1,1,0,stream>>>(
            overflow_flag.DATA_PTR<int>(),
            step_and_loss_scaler.DATA_PTR<double>());
}

void fused_maybe_adam_undo_cuda(
        at::Tensor & overflow_flag,
        at::Tensor & p,
        at::Tensor & m,
        at::Tensor & v,
        at::Tensor & g,
        float lr,
        float beta1,
        float beta2,
        float eps,
        float grad_scale,
        int step,
        int mode,
        int bias_correction,
        float decay)
{
    //Get tensor size
    int tsize = p.numel();
    //Determine #threads and #blocks
    const int threadsPerBlock = 512;
    const dim3 blocks((tsize+threadsPerBlock-1)/threadsPerBlock);
    AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p), "parameter tensor is too large to be indexed with int32");
    //Constants
    float step_size = 0;
    if (bias_correction == 1) {
        const float bias_correction1 = 1 - std::pow(beta1, step);
        const float bias_correction2 = 1 - std::pow(beta2, step);
        step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
    }
    else {
        step_size = lr;
    }
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    if (g.scalar_type() == at::ScalarType::Half) {
        //all other values should be fp32 for half gradients
        AT_ASSERTM(p.scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
        //dispatch is done on the gradient type
        using namespace at; // prevents "toString is undefined" errors
        DISPATCH_FLOAT_AND_HALF(g.scalar_type(), 0, "adam_cuda_kernel",
                using accscalar_t = at::acc_type<scalar_t_0, true>;
                maybe_adam_undo_cuda_kernel<accscalar_t, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                    overflow_flag.numel() ? overflow_flag.DATA_PTR<int>() : NULL,
                    p.DATA_PTR<accscalar_t>(),
                    m.DATA_PTR<accscalar_t>(),
                    v.DATA_PTR<accscalar_t>(),
                    g.DATA_PTR<scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    tsize,
                    (adamMode_t) mode,
                    decay);
                );
    } else {
        using namespace at;
        DISPATCH_DOUBLE_AND_FLOAT(g.scalar_type(), 0, "adam_cuda_kernel",
                maybe_adam_undo_cuda_kernel<scalar_t_0, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                    overflow_flag.numel() ? overflow_flag.DATA_PTR<int>() : NULL,
                    p.DATA_PTR<scalar_t_0>(),
                    m.DATA_PTR<scalar_t_0>(),
                    v.DATA_PTR<scalar_t_0>(),
                    g.DATA_PTR<scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    tsize,
                    (adamMode_t) mode,
                    decay);
                );
    }
    THCudaCheck(hipGetLastError());
}

void fused_adam_cuda_mt(
    int chunk_size,
    at::Tensor overflow_flag,
    std::vector<std::vector<at::Tensor>> tensor_lists, // p, m, v, g, p_copy
    float lr,
    float beta1,
    float beta2,
    float eps,
    float grad_scale,
    int step,
    int mode,
    int bias_correction,
    float decay) {

    //Constants
    float step_size = 0;
    if (bias_correction == 1) {
        const float bias_correction1 = 1 - std::pow(beta1, step);
        const float bias_correction2 = 1 - std::pow(beta2, step);
        step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
    }
    else {
        step_size = lr;
    }
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    size_t tl_sz = tensor_lists.size();
    AT_ASSERTM(tl_sz == 4 || tl_sz == 5, "expected tensor lists of size 4 or 5");

    if (tensor_lists[3][0].scalar_type() == at::ScalarType::Half) {
//alher values should be fp32 for half gradients
        AT_ASSERTM(tensor_lists[0][0].scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
//dich is done on the gradient type
        if (tl_sz == 5) {
            DISPATCH_FLOAT_AND_HALF(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                    using accscalar_t = at::acc_type<scalar_t_0, true>;
                    multi_tensor_apply<5>(
                        BLOCK_SIZE,
                        chunk_size,
                        overflow_flag,
                        tensor_lists,
                        AdamFunctor<5, accscalar_t, scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        (adamMode_t) mode,
                        decay);
                    );
        } else {
            DISPATCH_FLOAT_AND_HALF(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                    using accscalar_t = at::acc_type<scalar_t_0, true>;
                    multi_tensor_apply<4>(
                        BLOCK_SIZE,
                        chunk_size,
                        overflow_flag,
                        tensor_lists,
                        AdamFunctor<4, accscalar_t, scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        (adamMode_t) mode,
                        decay);
                    );
        }
    } else {
        if (tl_sz == 5) {
            DISPATCH_DOUBLE_AND_FLOAT(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                    multi_tensor_apply<5>(
                        BLOCK_SIZE,
                        chunk_size,
                        overflow_flag,
                        tensor_lists,
                        AdamFunctor<5, scalar_t_0, scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        (adamMode_t) mode,
                        decay);
                    );
        } else {
            DISPATCH_DOUBLE_AND_FLOAT(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                    multi_tensor_apply<4>(
                        BLOCK_SIZE,
                        chunk_size,
                        overflow_flag,
                        tensor_lists,
                        AdamFunctor<4, scalar_t_0, scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        (adamMode_t) mode,
                        decay);
                    );
        }
    }
    THCudaCheck(hipGetLastError());
}

void fused_maybe_adam_undo_cuda_mt(
    int chunk_size,
    at::Tensor overflow_flag,
    std::vector<std::vector<at::Tensor>> tensor_lists, // p, m, v, g, p_copy
    float lr,
    float beta1,
    float beta2,
    float eps,
    float grad_scale,
    int step,
    int mode,
    int bias_correction,
    float decay) {

    //Constants
    float step_size = 0;
    if (bias_correction == 1) {
        const float bias_correction1 = 1 - std::pow(beta1, step);
        const float bias_correction2 = 1 - std::pow(beta2, step);
        step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
    }
    else {
        step_size = lr;
    }
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    size_t tl_sz = tensor_lists.size();
    AT_ASSERTM(tl_sz == 4, "expected tensor list of size 4");

    if (tensor_lists[3][0].scalar_type() == at::ScalarType::Half) {
        //alher values should be fp32 for half gradients
        AT_ASSERTM(tensor_lists[0][0].scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
        //dich is done on the gradient type
        DISPATCH_FLOAT_AND_HALF(tensor_lists[3][0].scalar_type(), 0, "maybe_adam_undo_cuda_mt_kernel",
                using accscalar_t = at::acc_type<scalar_t_0, true>;
                multi_tensor_apply<4>(
                    BLOCK_SIZE,
                    chunk_size,
                    overflow_flag,
                    tensor_lists,
                    MaybeAdamUndoFunctor<4, accscalar_t, scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    (adamMode_t) mode,
                    decay);
                );
    } else {
        DISPATCH_DOUBLE_AND_FLOAT(tensor_lists[3][0].scalar_type(), 0, "maybe_adam_undo_cuda_mt_kernel",
                multi_tensor_apply<4>(
                    BLOCK_SIZE,
                    chunk_size,
                    overflow_flag,
                    tensor_lists,
                    MaybeAdamUndoFunctor<4, scalar_t_0, scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    (adamMode_t) mode,
                    decay);
                );
    }
    THCudaCheck(hipGetLastError());
}

