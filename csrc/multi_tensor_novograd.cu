#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

typedef enum{
  MOMENT_MODE_0   =0, // Momentum with denom/decay, optional grad averaging after
  MOMENT_MODE_1   =1  // Momentum without denom/decay
} momentMode_t;

void multi_tensor_norm_out_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor out,
  const float alpha,
  const float beta,
  const int norm_type);

template<typename T>
struct NovoGradFunctor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<3>& tl,
    const float beta1,
    const float beta2,
    const float beta3,
    const float eps,
    const float step_size,
    momentMode_t m_mode,
    const float decay,
    const float* per_tensor_grad_norm)
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int tensor_num = tl.start_tensor_this_launch + tensor_loc;
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    float grad_norm = per_tensor_grad_norm[tensor_num];

    T* g = (T*)tl.addresses[0][tensor_loc];
    g += chunk_idx*chunk_size;

    T* p = (T*)tl.addresses[1][tensor_loc];
    p += chunk_idx*chunk_size;

    T* m = (T*)tl.addresses[2][tensor_loc];
    m += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for(int i_start = 0;
            i_start < n && i_start < chunk_size;
            i_start += blockDim.x*ILP)
    {
      T r_g[ILP];
      T r_p[ILP];
      T r_m[ILP];
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          r_g[ii] = g[i];
          r_p[ii] = p[i];
          r_m[ii] = m[i];
        } else {
          r_g[ii] = T(0);
          r_p[ii] = T(0);
          r_m[ii] = T(0);
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        if (m_mode == MOMENT_MODE_0) {
          T denom = grad_norm + eps;
          r_g[ii] = (r_g[ii] / denom) + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + beta3 * r_g[ii];
          r_p[ii] = r_p[ii] - (step_size * r_m[ii]);
        }
        else {
          r_m[ii] = beta1 * r_m[ii] + beta3 * r_g[ii];
          T denom = grad_norm + eps;
          T update = (r_m[ii] / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (step_size * update);
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          p[i] = r_p[ii];
          m[i] = r_m[ii];
        }
      }
    }
  }
};

void multi_tensor_novograd_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor grad_norms,
  const float lr,
  const float beta1,
  const float beta2,
  const float epsilon,
  const int step,
  const int bias_correction,
  const float weight_decay,
  const int grad_averaging,
  const int moment_mode,
  const int norm_type)
{
  using namespace at;

  // Handle bias correction mode
  float step_size = 0;
  if (bias_correction == 1) {
    const float bias_correction1 = 1 - std::pow(beta1, step);
    const float bias_correction2 = 1 - std::pow(beta2, step);
    step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
  }
  else {
    step_size = lr;
  }

  // Handle grad averaging mode
  float beta3 = 1;
  if (grad_averaging == 1) beta3 = 1 - beta1;

  std::vector<std::vector<at::Tensor>> grad_list(tensor_lists.begin(), tensor_lists.begin()+1);

  // Compute and update grad norm
  // Here use a per tensor norm, and blend new norm(n) and old norm(gn) by
  // L-2: gn = sqrt(a * gn^2 + b * n^2)
  // L-inf: gn = a * gn + b * n
  multi_tensor_norm_out_cuda(chunk_size, noop_flag, grad_list, grad_norms, beta2, (1.0f - beta2), norm_type);

  // Assume single type across p,g,m1,m2 now
  DISPATCH_DOUBLE_FLOAT_AND_HALF(
    tensor_lists[0][0].scalar_type(), 0, "novograd",
    multi_tensor_apply<3>(
      BLOCK_SIZE,
      chunk_size,
      noop_flag,
      tensor_lists,
      NovoGradFunctor<scalar_t_0>(),
      beta1,
      beta2,
      beta3, // 1-beta1 or 1 depends on averaging mode
      epsilon,
      step_size,
      (momentMode_t) moment_mode,
      weight_decay,
      grad_norms.data<float>()); )

  AT_CUDA_CHECK(hipGetLastError());

}
