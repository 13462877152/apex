#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include "multi_tensor_apply.h"

#include <assert.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256
#define ILP 4

template<typename in_t>
struct UnscaleFunctor
{
  __device__ __forceinline__ void operator()(
    volatile int* noop_flag,
    int tensor_idx,
    int chunk_idx,
    int chunk_size,
    int n,
    void** addresses,
    int addresses_x,
    float scale)
  {
    __shared__ int noop;

    in_t* in = (in_t*)addresses[tensor_idx];
    in += chunk_idx*chunk_size;
   
    float* out = (float*)addresses[addresses_x + tensor_idx];
    out += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    // Non-divergent exit condition for the __syncthreads
    float incoming_vals[ILP];
    for(int i_start = 0;
        i_start < n && i_start < chunk_size;
        i_start += blockDim.x*ILP)
    {
      if(threadIdx.x == 0)
        noop = *noop_flag;

      __syncthreads();

      if(noop == 1)
        break;

      #pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        incoming_vals[ii] = 0;
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n)
          incoming_vals[ii] = static_cast<float>(in[i]);
      }

      #pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n)
          if(isfinite(incoming_vals[ii]))
            out[i] = incoming_vals[ii]*scale;
          else
            *noop_flag = 1; // Blindly fire off a write.  These will race but that's ok.
      }    // This is NOT guaranteed to be seen immediately by thread 0 on the next iteration.
    }      // I wonder if there's a way we can rig the short-circuiting with only one syncthreads.
  }        // It's possible we can just lean on the cache (no smem or syncs) and still be fast.
};


void multi_tensor_unscale_cuda(
  int nblocks,
  at::Tensor noop_flag,
  at::Tensor cpu_tensor_addresses,
  at::Tensor gpu_block_to_tensor,
  at::Tensor gpu_block_to_chunk,
  at::Tensor gpu_tensor_sizes,
  at::Tensor gpu_tensor_addresses,
  int chunk_size,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  float scale)
{
  using namespace at;

  int addresses_x = gpu_tensor_addresses.size(1);

  // <.< >.> i don't see any cops. i'm going to access the pointers directly.
  // auto addresses_a = cpu_tensor_addresses.accessor<int64_t, 2>();
  // This logic could be moved to prep_multi_tensor_launch, but we might need to
  // pick which kernel instantiation to launch based on the RTTI of tensor_lists,
  // so we may as well accept tensor_lists and extract the pointers here.
  void** addresses_a = (void**)cpu_tensor_addresses.data_ptr();

  int len0 = tensor_lists[0].size();
  for(unsigned int l = 0; l < tensor_lists.size(); l++)
  {
    AT_CHECK(tensor_lists[l].size() == len0, "Lengths of tensor lists do not match.");
    for(unsigned int t = 0; t < tensor_lists[l].size(); t++)
    {
      AT_CHECK(tensor_lists[l][t].numel() == tensor_lists[0][t].numel(),
        "Numel mismatch in corresponding tensors in different lists.");
      addresses_a[l*addresses_x + t] = tensor_lists[l][t].data_ptr();
      // addresses_a[l][t] = (void*)tensor_lists[l][t].data<float>();
    }
  }

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  gpu_tensor_addresses.copy_(cpu_tensor_addresses, 1/*non_blocking*/);
 
  // Lock the output (downscaled) type to float.
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(tensor_lists[0][0].type(),
     "multi_tensor_unscale_cuda",
     [&]
     {
       // using accscalar_t = acc_type<scalar_t, true>;
       multi_tensor_apply_kernel<<<nblocks, BLOCK_SIZE, 0, stream>>>(
         noop_flag.data<int>(),
         gpu_block_to_tensor.data<int>(),
         gpu_block_to_chunk.data<int>(),
         gpu_tensor_sizes.data<int>(),
         chunk_size,
         (void**)gpu_tensor_addresses.data_ptr(),
         addresses_x,
         UnscaleFunctor<scalar_t>(),
         scale);
     });

  AT_CUDA_CHECK(hipGetLastError());
  // AT_CUDA_CHECK(hipDeviceSynchronize());
}
