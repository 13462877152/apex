#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

typedef enum{
  ADAM_MODE_0   =0, // eps under square root
  ADAM_MODE_1   =1  // eps outside square root
} adamMode_t;


template<typename T>
struct AdamFunctor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<4>& tl,
    const float beta1,
    const float beta2,
    const float eps,
    const float step_size,
    adamMode_t mode,
    const float decay)
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];

    // potentially use to pass in list of scalar
    // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    T* g = (T*)tl.addresses[0][tensor_loc];
    g += chunk_idx*chunk_size;

    T* p = (T*)tl.addresses[1][tensor_loc];
    p += chunk_idx*chunk_size;

    T* m = (T*)tl.addresses[2][tensor_loc];
    m += chunk_idx*chunk_size;

    T* v = (T*)tl.addresses[3][tensor_loc];
    v += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for(int i_start = 0;
            i_start < n && i_start < chunk_size;
            i_start += blockDim.x*ILP)
    {
      T r_g[ILP];
      T r_p[ILP];
      T r_m[ILP];
      T r_v[ILP];
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          r_g[ii] = g[i];
          r_p[ii] = p[i];
          r_m[ii] = m[i];
          r_v[ii] = v[i];
        } else {
          r_g[ii] = T(0);
          r_p[ii] = T(0);
          r_m[ii] = T(0);
          r_v[ii] = T(0);
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        r_m[ii] = beta1 * r_m[ii] + (1-beta1) * r_g[ii];
        r_v[ii] = beta2 * r_v[ii] + (1-beta2) * r_g[ii] * r_g[ii];
        T denom;
        if (mode == ADAM_MODE_0)
          denom = sqrtf(r_v[ii] + eps);
        else // Mode 1
          denom = sqrtf(r_v[ii]) + eps;
        T update = (r_m[ii] / denom) + (decay * r_p[ii]);
        r_p[ii] = r_p[ii] - (step_size * update);
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          p[i] = r_p[ii];
          m[i] = r_m[ii];
          v[i] = r_v[ii];
        }
      }
    }
  }
};

void multi_tensor_adam_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  const float lr,
  const float beta1,
  const float beta2,
  const float epsilon,
  const int step,
  const int eps_mode,
  const int bias_correction,
  const float weight_decay)
{
  using namespace at;

  float step_size = 0;
  if (bias_correction == 1) {
    const float bias_correction1 = 1 - std::pow(beta1, step);
    const float bias_correction2 = 1 - std::pow(beta2, step);
    step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
  }
  else {
    step_size = lr;
  }

  // Assume single type across p,g,m1,m2 now
  DISPATCH_DOUBLE_FLOAT_AND_HALF(
    tensor_lists[0][0].scalar_type(), 0, "adam",
    multi_tensor_apply<4>(
      BLOCK_SIZE,
      chunk_size,
      noop_flag,
      tensor_lists,
      AdamFunctor<scalar_t_0>(),
      beta1,
      beta2,
      epsilon,
      step_size,
      (adamMode_t) eps_mode,
      weight_decay); )

  AT_CUDA_CHECK(hipGetLastError());

}
