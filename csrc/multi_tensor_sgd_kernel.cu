#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include "multi_tensor_apply.cuh"

#include <assert.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 512
#define ILP 4

/**
 * Perform fused SGD on multiple buffers
 * N: number of tensors
 * tl[0] : gradients
 * tl[1] : weights
 * tl[2] : momentum buffers
 * tl[3] : fp16 weights (if appropriate)
 * wd : weight_decay (scalar)
 * momentum : momentum (scalar)
 * dampening : momentum dampening (scalar)
 * lr : learning rate (scalar)
 * nesterov : enable nesterov (bool)
 * first run : necessary for proper momentum handling & init
 **/
template<int N, typename T_grad, typename T_weight>
struct SGDFunctor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorList<N>& tl,
    float wd,
    float momentum,
    float dampening,
    float lr,
    bool nesterov,
    bool first_run)
  {
    __shared__ int noop_smem;

    if(threadIdx.x == 0)
      noop_smem = *noop_gmem;
    __syncthreads();
    if(noop_smem == 1)
      return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    T_grad* grad_in = (T_grad*)tl.addresses[0][tensor_loc];
    grad_in += chunk_idx*chunk_size;
   
    T_weight* weight_in = (T_weight*)tl.addresses[1][tensor_loc];
    weight_in += chunk_idx*chunk_size;

    T_weight* mom_in = (T_weight*)tl.addresses[2][tensor_loc];
    mom_in += chunk_idx*chunk_size;

    at::Half *model_weights_out = nullptr;
    if (N == 4) {
      model_weights_out = (at::Half*)tl.addresses[3][tensor_loc];
      model_weights_out += chunk_idx*chunk_size;
    }

    n -= chunk_idx*chunk_size;

    // Non-divergent exit condition for the __syncthreads
    float incoming_grads[ILP];
    float incoming_weights[ILP];
    float incoming_moms[ILP];
    for(int i_start = 0;
        i_start < n && i_start < chunk_size;
        i_start += blockDim.x*ILP)
    {
      #pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        incoming_grads[ii] = 0;
        incoming_weights[ii] = 0;
        incoming_moms[ii] = 0;
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
          incoming_grads[ii] = static_cast<float>(grad_in[i]);
          incoming_weights[ii] = static_cast<float>(weight_in[i]);
          incoming_moms[ii] = static_cast<float>(mom_in[i]);
      }

      // note for clarification to future michael:
      // From a pure memory dependency perspective, there's likely no point unrolling
      // the write loop, since writes just fire off once their LDGs arrive.
      // Put another way, the STGs are dependent on the LDGs, but not on each other.
      // There is still compute ILP benefit from unrolling the loop though.
      #pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size) {
          // apply weight decay
          if (wd != 0.f) {
            incoming_grads[ii] += wd * incoming_weights[ii];
          }
          if (momentum != 0.f) {
            if (!first_run) {
              incoming_moms[ii] = incoming_moms[ii] * momentum + (1.f - dampening) * incoming_grads[ii];
            }

            if (nesterov) {
              incoming_grads[ii] += momentum * incoming_moms[ii];
            }
          }

          // adjust the weight and write out
          weight_in[i] += (-lr * incoming_grads[ii]);

          // if necessary, write out an fp16 copy of the weights
          if (N == 4) {
            model_weights_out[i] = static_cast<at::Half>(weight_in[i]);
          }

          // also write out the new momentum
          if (momentum != 0.f) {
            mom_in[i] = incoming_moms[ii];
          }
        }
      }

      // *noop_gmem = 1 is NOT guaranteed to be seen immediately by thread 0.  I wonder if
      // we can rig block-wide and grid-wide short-circuiting with only one syncthreads.
      // It's possible we can just lean on the cache (no smem or syncs) and still be fast.
      if(threadIdx.x == 0)
        noop_smem = *noop_gmem;
      __syncthreads();
      if(noop_smem == 1)
        break;
    }
  }
};

void multi_tensor_sgd_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  float wd,
  float momentum,
  float dampening,
  float lr,
  bool nesterov,
  bool first_run)
{
  auto num_tensors = tensor_lists.size();
  auto grad_type = tensor_lists[0][0].type().scalarType();
  auto weight_type = tensor_lists[0][0].type().scalarType();

  // We have 4 potentials to handle here, in terms of
  // grad_type, param_type, momentum_type, requires_fp16_copy
  // 1. fp16, fp16, fp16, No
  // 2. fp16, fp32, fp32, No
  // 3. fp16, fp32, fp32, Yes
  // 4. fp32, fp32, fp32, No
  // It's easier to hardcode these possibilities than to use
  // switches etc. to handle the cross-product of cases where
  // we don't want the majority of them.

  // Case 1. fp16, fp16, fp16, No
  if (grad_type == at::ScalarType::Half &&
      weight_type == at::ScalarType::Half &&
      num_tensors == 3) {
    multi_tensor_apply<3>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        SGDFunctor<3, at::Half, at::Half>(),
        wd,
        momentum,
        dampening,
        lr,
        nesterov,
        first_run);
  }
  // Case 2. fp16, fp32, fp32, No
  else if (grad_type == at::ScalarType::Half &&
           weight_type == at::ScalarType::Float &&
           num_tensors == 3) {
    multi_tensor_apply<3>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        SGDFunctor<3, at::Half, float>(),
        wd,
        momentum,
        dampening,
        lr,
        nesterov,
        first_run);
  }
  // Case 3. fp16, fp32, fp32, Yes
  else if (grad_type == at::ScalarType::Half &&
           weight_type == at::ScalarType::Float &&
           num_tensors == 4) {
    multi_tensor_apply<4>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        SGDFunctor<4, at::Half, float>(),
        wd,
        momentum,
        dampening,
        lr,
        nesterov,
        first_run);
  }
  // Case 4. fp32, fp32, fp32, No
  else if (grad_type == at::ScalarType::Float &&
      weight_type == at::ScalarType::Float &&
      num_tensors == 3) {
    multi_tensor_apply<3>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        SGDFunctor<3, float, float>(),
        wd,
        momentum,
        dampening,
        lr,
        nesterov,
        first_run);
  }
  else {
    AT_ERROR("multi_tensor_sgd only supports some combinations of gradient & weight types. Given: ",
             "gradient: ", grad_type, ", weight: ", weight_type, ", num_lists: ", num_tensors);
  }

  AT_CUDA_CHECK(hipGetLastError());
}
